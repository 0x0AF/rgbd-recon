#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <GL/gl.h>
#include <GL/glext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cuda_gl_interop.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <reconstruction/cuda/glm.cuh>
#include <reconstruction/cuda/resources.cuh>
#include <reconstruction/cuda/util.cuh>

#include <reconstruction/cuda/copy_reference.cuh>

#include <reconstruction/cuda/ed_sample.cuh>

#include <reconstruction/cuda/pcg_solve.cuh>

#include <reconstruction/cuda/fuse_data.cuh>

extern "C" void init_cuda(glm::uvec3 &volume_res, struct_native_handles &native_handles)
{
    hipDeviceReset();

    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        if(deviceProperties.major >= 6 && deviceProperties.minor >= 1)
        {
            hipSetDevice(deviceIndex);
            hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
        }
    }

    size_t value;

    hipDeviceGetLimit(&value, hipLimitMallocHeapSize);
    printf("\n\nLimit Malloc Heap Size: %lu\n", value);

    hipDeviceGetLimit(&value, hipLimitStackSize);
    printf("\nLimit Stack Size: %lu\n", value);

    hipDeviceGetLimit(&value, cudaLimitDevRuntimeSyncDepth);
    printf("\nLimit Sync Depth: %lu\n", value);

    hipDeviceGetLimit(&value, cudaLimitDevRuntimePendingLaunchCount);
    printf("\nLimit Pending Launch: %lu\n\n", value);

    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_vertex_counter, native_handles.buffer_vertex_counter, hipGraphicsRegisterFlagsReadOnly));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_reference_mesh_vertices, native_handles.buffer_reference_vertices, hipGraphicsRegisterFlagsReadOnly));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_bricks, native_handles.buffer_bricks, hipGraphicsRegisterFlagsReadOnly));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_occupied, native_handles.buffer_occupied, hipGraphicsRegisterFlagsReadOnly));
    checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.volume_tsdf_data, native_handles.volume_tsdf_data, GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.array2d_kinect_depths, native_handles.array2d_kinect_depths, GL_TEXTURE_2D_ARRAY_EXT, hipGraphicsRegisterFlagsReadOnly));

    for(unsigned int i = 0; i < 4; i++)
    {
        checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.volume_cv_xyz_inv[i], native_handles.volume_cv_xyz_inv[i], GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    }

    hipExtent volume_extent = make_hipExtent(volume_res.x, volume_res.y, volume_res.z);
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors(hipMalloc3DArray(&_volume_array_tsdf_ref, &channel_desc, volume_extent, hipArraySurfaceLoadStore));

    hipblasCreate(&cublas_handle);
    getLastCudaError("hipblasCreate failure");

    hipsparseCreate(&cusparse_handle);
    getLastCudaError("hipsparseCreate failure");
}

extern "C" void deinit_cuda()
{
    hipsparseDestroy(cusparse_handle);
    hipblasDestroy(cublas_handle);

    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_vertex_counter));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_reference_mesh_vertices));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_bricks));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_occupied));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.volume_tsdf_data));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.array2d_kinect_depths));

    for(unsigned int i = 0; i < 4; i++)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(_cgr.volume_cv_xyz_inv[i]));
    }

    if(_volume_array_tsdf_ref != nullptr)
    {
        checkCudaErrors(hipFree(_volume_array_tsdf_ref));
    }

    if(_ed_graph != nullptr)
    {
        checkCudaErrors(hipFree(_ed_graph));
    }

    if(_jtf != nullptr)
    {
        checkCudaErrors(hipFree(_jtf));
    }

  if(_jtj_vals != nullptr)
    {
      checkCudaErrors(hipFree(_jtj_vals));
    }

  if(_jtj_rows != nullptr)
    {
      checkCudaErrors(hipFree(_jtj_rows));
    }

  if(_jtj_cols != nullptr)
    {
      checkCudaErrors(hipFree(_jtj_cols));
    }

    if(_h != nullptr)
    {
        checkCudaErrors(hipFree(_h));
    }

    if(pcg_Ax != nullptr)
    {
        checkCudaErrors(hipFree(pcg_Ax));
    }

    if(pcg_omega != nullptr)
    {
        checkCudaErrors(hipFree(pcg_omega));
    }

    if(pcg_p != nullptr)
    {
        checkCudaErrors(hipFree(pcg_p));
    }

    hipDeviceReset();
}
