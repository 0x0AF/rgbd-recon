#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <GL/gl.h>
#include <GL/glext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hipblas.h>
#include <cuda_gl_interop.h>
#include <hipsparse.h>

#include <reconstruction/cuda/glm.cuh>
#include <reconstruction/cuda/resources.cuh>
#include <reconstruction/cuda/util.cuh>

#include <reconstruction/cuda/copy_reference.cuh>
#include <reconstruction/cuda/ed_sample.cuh>
#include <reconstruction/cuda/fuse_data.cuh>
#include <reconstruction/cuda/hull.cuh>
#include <reconstruction/cuda/pcg_solve.cuh>
#include <reconstruction/cuda/sift.cuh>

extern "C" void init_cuda(glm::uvec3 &volume_res, struct_measures &measures, struct_native_handles &native_handles)
{
    hipDeviceReset();

    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        if(deviceProperties.major >= 6 && deviceProperties.minor >= 1)
        {
            hipSetDevice(deviceIndex);
            hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
            hipDeviceSetCacheConfig(hipFuncCachePreferShared);
            hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
        }
    }

    size_t value;

    hipDeviceGetLimit(&value, hipLimitMallocHeapSize);
    printf("\n\nLimit Malloc Heap Size: %lu\n", value);

    hipDeviceGetLimit(&value, hipLimitStackSize);
    printf("\nLimit Stack Size: %lu\n", value);

    hipDeviceGetLimit(&value, cudaLimitDevRuntimeSyncDepth);
    printf("\nLimit Sync Depth: %lu\n", value);

    hipDeviceGetLimit(&value, cudaLimitDevRuntimePendingLaunchCount);
    printf("\nLimit Pending Launch: %lu\n\n", value);

    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_vertex_counter, native_handles.buffer_vertex_counter, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_reference_mesh_vertices, native_handles.buffer_reference_vertices, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_bricks, native_handles.buffer_bricks, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_occupied, native_handles.buffer_occupied, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_ed_nodes_debug, native_handles.buffer_ed_nodes_debug, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_sorted_vertices_debug, native_handles.buffer_sorted_vertices_debug, hipGraphicsRegisterFlagsNone));

    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.texture_kinect_rgbs, native_handles.texture_kinect_rgbs, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.texture_kinect_depths, native_handles.texture_kinect_depths, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.texture_kinect_silhouettes, native_handles.texture_kinect_silhouettes, hipGraphicsRegisterFlagsNone));

    for(unsigned int i = 0; i < 4; i++)
    {
        checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.volume_cv_xyz_inv[i], native_handles.volume_cv_xyz_inv[i], GL_TEXTURE_3D, hipGraphicsRegisterFlagsNone));
        checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.volume_cv_xyz[i], native_handles.volume_cv_xyz[i], GL_TEXTURE_3D, hipGraphicsRegisterFlagsNone));
    }

    checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.volume_tsdf_data, native_handles.volume_tsdf_data, GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.volume_tsdf_ref, native_handles.volume_tsdf_ref, GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));

    memcpy(&_host_res.measures, &measures, sizeof(struct_measures));

    checkCudaErrors(hipMalloc(&_dev_res.kinect_rgbs, _host_res.measures.color_res.x * _host_res.measures.color_res.y * 4 * sizeof(float4)));
    checkCudaErrors(hipMalloc(&_dev_res.kinect_depths, _host_res.measures.depth_res.x * _host_res.measures.depth_res.y * 4 * sizeof(float2)));
    checkCudaErrors(hipMalloc(&_dev_res.kinect_silhouettes, _host_res.measures.depth_res.x * _host_res.measures.depth_res.y * 4 * sizeof(float1)));

    _host_res.kernel_gauss = (float *)malloc(KERNEL_LENGTH * sizeof(float));

    _host_res.kernel_gauss[0] = 0.012318f;
    _host_res.kernel_gauss[1] = 0.014381f;
    _host_res.kernel_gauss[2] = 0.016624f;
    _host_res.kernel_gauss[3] = 0.019024f;
    _host_res.kernel_gauss[4] = 0.021555f;
    _host_res.kernel_gauss[5] = 0.02418f;
    _host_res.kernel_gauss[6] = 0.026854f;
    _host_res.kernel_gauss[7] = 0.029528f;
    _host_res.kernel_gauss[8] = 0.032145f;
    _host_res.kernel_gauss[9] = 0.034647f;
    _host_res.kernel_gauss[10] = 0.036972f;
    _host_res.kernel_gauss[11] = 0.03906f;
    _host_res.kernel_gauss[12] = 0.040857f;
    _host_res.kernel_gauss[13] = 0.042311f;
    _host_res.kernel_gauss[14] = 0.043381f;
    _host_res.kernel_gauss[15] = 0.044036f;
    _host_res.kernel_gauss[16] = 0.044256f;
    _host_res.kernel_gauss[17] = 0.044036f;
    _host_res.kernel_gauss[18] = 0.043381f;
    _host_res.kernel_gauss[19] = 0.042311f;
    _host_res.kernel_gauss[20] = 0.040857f;
    _host_res.kernel_gauss[21] = 0.03906f;
    _host_res.kernel_gauss[22] = 0.036972f;
    _host_res.kernel_gauss[23] = 0.034647f;
    _host_res.kernel_gauss[24] = 0.032145f;
    _host_res.kernel_gauss[25] = 0.029528f;
    _host_res.kernel_gauss[26] = 0.026854f;
    _host_res.kernel_gauss[27] = 0.02418f;
    _host_res.kernel_gauss[28] = 0.021555f;
    _host_res.kernel_gauss[29] = 0.019024f;
    _host_res.kernel_gauss[30] = 0.016624f;
    _host_res.kernel_gauss[31] = 0.014381f;
    _host_res.kernel_gauss[32] = 0.012318f;

    hipblasCreate(&cublas_handle);
    getLastCudaError("hipblasCreate failure");

    hipsparseCreate(&cusparse_handle);
    getLastCudaError("hipsparseCreate failure");

    hipsolverSpCreate(&cusolver_handle);
    getLastCudaError("hipsolverSpCreate failure");
}

extern "C" void deinit_cuda()
{
    free(_host_res.kernel_gauss);

    hipsparseDestroy(cusparse_handle);
    hipblasDestroy(cublas_handle);
    hipsolverSpDestroy(cusolver_handle);

    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_vertex_counter));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_reference_mesh_vertices));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_bricks));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_occupied));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_ed_nodes_debug));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_sorted_vertices_debug));

    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.volume_tsdf_data));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.volume_tsdf_ref));

    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.texture_kinect_rgbs));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.texture_kinect_depths));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.texture_kinect_silhouettes));

    for(unsigned int i = 0; i < 4; i++)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(_cgr.volume_cv_xyz_inv[i]));
        checkCudaErrors(hipGraphicsUnregisterResource(_cgr.volume_cv_xyz[i]));
    }

    if(_dev_res.kinect_rgbs != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.kinect_rgbs));
    }

    if(_dev_res.kinect_depths != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.kinect_depths));
    }

    if(_dev_res.kinect_silhouettes != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.kinect_silhouettes));
    }

    if(_dev_res.ed_graph != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.ed_graph));
    }

    if(_dev_res.jtf != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.jtf));
    }

    if(_dev_res.jtj_vals != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.jtj_vals));
    }

    if(_dev_res.jtj_rows != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.jtj_rows));
    }

    if(_dev_res.jtj_cols != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.jtj_cols));
    }

    if(_dev_res.h != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.h));
    }

    if(_dev_res.pcg_Ax != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.pcg_Ax));
    }

    if(_dev_res.pcg_omega != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.pcg_omega));
    }

    if(_dev_res.pcg_p != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.pcg_p));
    }

    hipDeviceReset();
}
