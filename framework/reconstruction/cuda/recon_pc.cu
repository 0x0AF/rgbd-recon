#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <GL/gl.h>
#include <GL/glext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hipblas.h>
#include <cuda_gl_interop.h>
#include <hipsparse.h>

#include <reconstruction/cuda/glm.cuh>
#include <reconstruction/cuda/resources.cuh>
#include <reconstruction/cuda/util.cuh>

#include <reconstruction/cuda/copy_reference.cuh>
#include <reconstruction/cuda/ed_sample.cuh>
#include <reconstruction/cuda/fuse_data.cuh>
#include <reconstruction/cuda/pcg_solve.cuh>
#include <reconstruction/cuda/sift.cuh>

extern "C" void init_cuda(glm::uvec3 &volume_res, struct_measures &measures, struct_native_handles &native_handles)
{
    hipDeviceReset();

    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        if(deviceProperties.major >= 6 && deviceProperties.minor >= 1)
        {
            hipSetDevice(deviceIndex);
            hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
            hipDeviceSetCacheConfig(hipFuncCachePreferShared);
            hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
        }
    }

    size_t value;

    hipDeviceGetLimit(&value, hipLimitMallocHeapSize);
    printf("\n\nLimit Malloc Heap Size: %lu\n", value);

    hipDeviceGetLimit(&value, hipLimitStackSize);
    printf("\nLimit Stack Size: %lu\n", value);

    hipDeviceGetLimit(&value, cudaLimitDevRuntimeSyncDepth);
    printf("\nLimit Sync Depth: %lu\n", value);

    hipDeviceGetLimit(&value, cudaLimitDevRuntimePendingLaunchCount);
    printf("\nLimit Pending Launch: %lu\n\n", value);

    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_vertex_counter, native_handles.buffer_vertex_counter, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_reference_mesh_vertices, native_handles.buffer_reference_vertices, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_bricks, native_handles.buffer_bricks, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_occupied, native_handles.buffer_occupied, hipGraphicsRegisterFlagsNone));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&_cgr.buffer_ed_nodes_debug, native_handles.buffer_ed_nodes_debug, hipGraphicsRegisterFlagsNone));

    // TODO: rgbs output
    /*checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.texture_kinect_rgbs, native_handles.texture_kinect_rgbs,GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));*/
    checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.texture_kinect_depths, native_handles.texture_kinect_depths, GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.texture_kinect_silhouettes, native_handles.texture_kinect_silhouettes, GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));

    for(unsigned int i = 0; i < 4; i++)
    {
        checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.volume_cv_xyz_inv[i], native_handles.volume_cv_xyz_inv[i], GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));
        checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.volume_cv_xyz[i], native_handles.volume_cv_xyz[i], GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    }

    checkCudaErrors(hipGraphicsGLRegisterImage(&_cgr.volume_tsdf_data, native_handles.volume_tsdf_data, GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));

    memcpy(&_host_res.measures, &measures, sizeof(struct_measures));

    hipExtent volume_extent = make_hipExtent(volume_res.x, volume_res.y, volume_res.z);
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors(hipMalloc3DArray(&_dev_res.volume_array_tsdf_ref, &channel_desc, volume_extent, hipArraySurfaceLoadStore));

    hipblasCreate(&cublas_handle);
    getLastCudaError("hipblasCreate failure");

    hipsparseCreate(&cusparse_handle);
    getLastCudaError("hipsparseCreate failure");

    hipsolverSpCreate(&cusolver_handle);
    getLastCudaError("hipsolverSpCreate failure");
}

extern "C" void deinit_cuda()
{
    hipsparseDestroy(cusparse_handle);
    hipblasDestroy(cublas_handle);
    hipsolverSpDestroy(cusolver_handle);

    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_vertex_counter));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_reference_mesh_vertices));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_bricks));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_occupied));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.buffer_ed_nodes_debug));
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.volume_tsdf_data));

    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.texture_kinect_depths));
    // TODO: rgbs output
    /*checkCudaErrors(hipGraphicsUnregisterResource(_cgr.texture_kinect_rgbs));*/
    checkCudaErrors(hipGraphicsUnregisterResource(_cgr.texture_kinect_silhouettes));

    for(unsigned int i = 0; i < 4; i++)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(_cgr.volume_cv_xyz_inv[i]));
        checkCudaErrors(hipGraphicsUnregisterResource(_cgr.volume_cv_xyz[i]));
    }

    if(_dev_res.volume_array_tsdf_ref != nullptr)
    {
        checkCudaErrors(hipFreeArray(_dev_res.volume_array_tsdf_ref));
    }

    if(_dev_res.ed_graph != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.ed_graph));
    }

    if(_dev_res.jtf != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.jtf));
    }

    if(_dev_res.jtj_vals != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.jtj_vals));
    }

    if(_dev_res.jtj_rows != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.jtj_rows));
    }

    if(_dev_res.jtj_cols != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.jtj_cols));
    }

    if(_dev_res.h != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.h));
    }

    if(_dev_res.pcg_Ax != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.pcg_Ax));
    }

    if(_dev_res.pcg_omega != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.pcg_omega));
    }

    if(_dev_res.pcg_p != nullptr)
    {
        checkCudaErrors(hipFree(_dev_res.pcg_p));
    }

    hipDeviceReset();
}
