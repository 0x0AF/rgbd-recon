#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <GL/gl.h>
#include <GL/glext.h>

#define GLM_ENABLE_EXPERIMENTAL

#include <glm/common.hpp>
#include <glm/exponential.hpp>
#include <glm/geometric.hpp>
#include <glm/integer.hpp>
#include <glm/mat2x2.hpp>
#include <glm/mat2x3.hpp>
#include <glm/mat2x4.hpp>
#include <glm/mat3x2.hpp>
#include <glm/mat3x3.hpp>
#include <glm/mat3x4.hpp>
#include <glm/mat4x2.hpp>
#include <glm/mat4x3.hpp>
#include <glm/mat4x4.hpp>
#include <glm/matrix.hpp>
#include <glm/packing.hpp>
#include <glm/trigonometric.hpp>
#include <glm/vec2.hpp>
#include <glm/vec3.hpp>
#include <glm/vec4.hpp>
#include <glm/vector_relational.hpp>

#include <glm/gtx/norm.hpp>

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_cusolver.h>
#include <hip/hip_vector_types.h>

#include <hipblas.h>
#include <hipsolver.h>

#include <cuda_occupancy.h>
#include <>
#include <hip/driver_types.h>
#include <iostream>
#include <texture_types.h>
#include <hip/hip_vector_types.h>

#include <recon_pc.cuh>

hipGraphicsResource *cgr_buffer_reference_mesh_vertices = nullptr;
hipGraphicsResource *cgr_buffer_vertex_counter = nullptr;
hipGraphicsResource *cgr_buffer_bricks = nullptr;
hipGraphicsResource *cgr_buffer_occupied = nullptr;
hipGraphicsResource *cgr_volume_tsdf_data = nullptr;
hipGraphicsResource *cgr_array2d_kinect_depths = nullptr;

surface<void, cudaSurfaceType3D> _volume_tsdf_data;
surface<void, cudaSurfaceType3D> _volume_tsdf_ref;

surface<void, cudaSurfaceType2DLayered> _array2d_kinect_depths;

hipExtent _volume_res;
struct_native_handles _native_handles;

hipArray *_volume_array_tsdf_ref = nullptr;
struct_ed_node *_ed_graph = nullptr;
float *_jtj = nullptr;
float *_jtf = nullptr;
float *_h = nullptr;

const unsigned ED_GRAPH_NODE_RES = 9u;
const unsigned BRICK_RES = 9u;
const unsigned BRICK_VOXEL_DIM = 6u;
const unsigned BRICK_VOXELS = 216u;
const unsigned VOLUME_VOXEL_DIM = 50u;

__global__ void kernel_copy_reference(GLuint *occupied_bricks, size_t occupied_brick_count)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < occupied_brick_count)
    {
        unsigned int brick_id = occupied_bricks[idx];

        if(brick_id == 0u)
        {
            return;
        }

        glm::uvec3 brick = glm::uvec3(0u);
        brick.z = brick_id / (BRICK_RES * BRICK_RES);
        brick_id %= (BRICK_RES * BRICK_RES);
        brick.y = brick_id / BRICK_RES;
        brick_id %= BRICK_RES;
        brick.x = brick_id;

        // printf("\nbrick %u: (%u,%u,%u)\n", brick_id, brick.x, brick.y, brick.z);

        for(unsigned int i = 0u; i < BRICK_VOXELS; i++)
        {
            unsigned int position_id = i;

            glm::uvec3 position = glm::uvec3(0u);
            position.z = position_id / (BRICK_VOXEL_DIM * BRICK_VOXEL_DIM);
            position_id %= (BRICK_VOXEL_DIM * BRICK_VOXEL_DIM);
            position.y = position_id / BRICK_VOXEL_DIM;
            position_id %= (BRICK_VOXEL_DIM);
            position.x = position_id;

            glm::uvec3 world = brick * BRICK_VOXEL_DIM + position;

            if(world.x >= VOLUME_VOXEL_DIM || world.y >= VOLUME_VOXEL_DIM || world.z >= VOLUME_VOXEL_DIM)
            {
                continue;
            }

            // printf("\nbrick %u, position %u: (%u,%u,%u)\n", occupied_bricks[idx], i, world.x, world.y, world.z);

            float2 data;
            surf3Dread(&data, _volume_tsdf_data, world.x * sizeof(float2), world.y, world.z);
            surf3Dwrite(data, _volume_tsdf_ref, world.x * sizeof(float2), world.y, world.z);
        }
    }
}

__global__ void kernel_sample_ed_nodes(GLuint *vx_counter, struct_vertex *vx_ptr, struct_ed_node *_ed_graph)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    long int u = x * VOLUME_VOXEL_DIM / ED_GRAPH_NODE_RES;
    long int v = y * VOLUME_VOXEL_DIM / ED_GRAPH_NODE_RES;
    long int d = z * VOLUME_VOXEL_DIM / ED_GRAPH_NODE_RES;

    long int node_id = x * ED_GRAPH_NODE_RES * ED_GRAPH_NODE_RES + y * ED_GRAPH_NODE_RES + z;

    GLuint counter = vx_counter[0];

    // printf("\ncounter: %u\n", counter);

    struct_ed_node *node = _ed_graph + node_id;

    for(long int i = 0; i < (long int)counter; i++)
    {
        glm::vec3 pos = vx_ptr[i].position;
        pos *= VOLUME_VOXEL_DIM;
        if(pos.x > u && pos.x < u + ED_GRAPH_NODE_RES && pos.y > v && pos.y < v + ED_GRAPH_NODE_RES && pos.z > d && pos.z < d + ED_GRAPH_NODE_RES)
        {
            node->position = pos;
            node->set = true;
            // printf("\nfound node: %f,%f,%f at %lu\n", pos.x, pos.y, pos.z, i);
            break;
        }
    }
}

__device__ glm::vec3 warp_position(glm::vec3 &pos, struct_ed_node &ed_node)
{
    pos *= VOLUME_VOXEL_DIM;

    glm::vec3 dist = pos - ed_node.position;
    float skinning_weight = expf(glm::length(dist) * glm::length(dist) * 2 / (ED_GRAPH_NODE_RES * ED_GRAPH_NODE_RES));
    return skinning_weight * (ed_node.affine * dist + ed_node.position + ed_node.translation);
}

__device__ glm::vec3 warp_normal(glm::vec3 &pos, glm::vec3 &normal, struct_ed_node &ed_node)
{
    pos *= VOLUME_VOXEL_DIM;

    glm::vec3 dist = pos - ed_node.position;
    float skinning_weight = expf(glm::length(dist) * glm::length(dist) * 2 / (ED_GRAPH_NODE_RES * ED_GRAPH_NODE_RES));
    return skinning_weight * (glm::transpose(glm::inverse(ed_node.affine)) * normal);
}

__device__ float evaluate_vx_residual(struct_vertex &vertex, struct_ed_node &ed_node)
{
    glm::vec3 warped_position = warp_position(vertex.position, ed_node);
    glm::vec3 warped_normal = warp_normal(vertex.position, vertex.normal, ed_node);

    float residuals = 0.f;

    for(int i = 0; i < 5; i++)
    {
        // TODO: lookup depth maps and extract projective term

        glm::vec3 extracted_position = warped_position;
        extracted_position *= (1 + 0.1 * fracf(sinf(warped_position.x)));

        residuals += glm::dot(warped_normal, warped_position - extracted_position);
    }

    return residuals;
}

__device__ float evaluate_vx_pd(struct_vertex &vertex, struct_ed_node &ed_node, int partial_derivative_index, float vx_residual)
{
    switch(partial_derivative_index)
    {
    case 0:
        ed_node.position.x += 0.035f;
        break;
    case 1:
        ed_node.position.y += 0.035f;
        break;
    case 2:
        ed_node.position.z += 0.035f;
        break;
    case 3:
        ed_node.affine[0][0] += 0.035f;
        break;
    case 4:
        ed_node.affine[0][1] += 0.035f;
        break;
    case 5:
        ed_node.affine[0][2] += 0.035f;
        break;
    case 6:
        ed_node.affine[1][0] += 0.035f;
        break;
    case 7:
        ed_node.affine[1][1] += 0.035f;
        break;
    case 8:
        ed_node.affine[1][2] += 0.035f;
        break;
    case 9:
        ed_node.affine[2][0] += 0.035f;
        break;
    case 10:
        ed_node.affine[2][1] += 0.035f;
        break;
    case 11:
        ed_node.affine[2][2] += 0.035f;
        break;
    case 12:
        ed_node.translation.x += 0.035f;
        break;
    case 13:
        ed_node.translation.y += 0.035f;
        break;
    case 14:
        ed_node.translation.z += 0.035f;
        break;
    }

    float residual_pos = evaluate_vx_residual(vertex, ed_node);

    switch(partial_derivative_index)
    {
    case 0:
        ed_node.position.x -= 0.035f;
        break;
    case 1:
        ed_node.position.y -= 0.035f;
        break;
    case 2:
        ed_node.position.z -= 0.035f;
        break;
    case 3:
        ed_node.affine[0][0] -= 0.035f;
        break;
    case 4:
        ed_node.affine[0][1] -= 0.035f;
        break;
    case 5:
        ed_node.affine[0][2] -= 0.035f;
        break;
    case 6:
        ed_node.affine[1][0] -= 0.035f;
        break;
    case 7:
        ed_node.affine[1][1] -= 0.035f;
        break;
    case 8:
        ed_node.affine[1][2] -= 0.035f;
        break;
    case 9:
        ed_node.affine[2][0] -= 0.035f;
        break;
    case 10:
        ed_node.affine[2][1] -= 0.035f;
        break;
    case 11:
        ed_node.affine[2][2] -= 0.035f;
        break;
    case 12:
        ed_node.translation.x -= 0.035f;
        break;
    case 13:
        ed_node.translation.y -= 0.035f;
        break;
    case 14:
        ed_node.translation.z -= 0.035f;
        break;
    }

    return (residual_pos - vx_residual) / 0.07f;
}

__device__ float *evaluate_ed_node_residuals(hipExtent &volume_res, struct_ed_node &ed_node)
{
    float *residuals = new float[2];

    glm::mat3 mat_1 = (glm::transpose(ed_node.affine) * ed_node.affine - glm::mat3());

    residuals[0] = 0.f;

    for(int i = 0; i < 3; i++)
    {
        for(int k = 0; k < 3; k++)
        {
            residuals[0] += mat_1[i][k] * mat_1[i][k];
        }
    }

    residuals[0] = (float)sqrt(residuals[0]);
    residuals[0] += glm::determinant(ed_node.affine) - 1;

    // TODO: figure out smooth component
    residuals[1] = 0.f;

    return residuals;
}

__global__ void kernel_jtj_jtf(float *jtj, float *jtf, GLuint *vx_counter, struct_vertex *vx_ptr, struct_ed_node *_ed_graph)
{
    const unsigned long long int block_id = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    const unsigned long long int thread_id = block_id * blockDim.x + threadIdx.x;

    GLuint vertex_counter = vx_counter[0];

    if(thread_id >= vertex_counter)
    {
        return;
    }

    struct_vertex vx = vx_ptr[thread_id];

    glm::vec3 pos = vx.position;
    pos *= VOLUME_VOXEL_DIM;

    long int node_id = (int)pos.x * ED_GRAPH_NODE_RES + (int)pos.y + (int)pos.z / ED_GRAPH_NODE_RES;
    struct_ed_node node = *(_ed_graph + node_id);

    float vx_residual = evaluate_vx_residual(vx, node);
    float *vx_pds = (float *)malloc(sizeof(float) * 15);

    for(int i = 0; i < 15; i++)
    {
        vx_pds[i] = evaluate_vx_pd(vx, node, i, vx_residual);
    }

    for(int i = 0; i < 15; i++)
    {
        jtf[node_id * 15 + i] += vx_pds[i] * vx_residual;

        for(int k = 0; k < 15; k++)
        {
            jtj[(node_id * 15 + i) * ED_GRAPH_NODE_RES + node_id * 15 + k] += vx_pds[i] * vx_pds[k];
        }
    }

    free(vx_pds);
}

__global__ void kernel_fuse_volume(GLuint *occupied_bricks, size_t occupied_brick_count)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < occupied_brick_count)
    {
        unsigned int brick_id = occupied_bricks[idx];

        if(brick_id == 0u)
        {
            return;
        }

        glm::uvec3 brick = glm::uvec3(0u);
        brick.z = brick_id / (BRICK_RES * BRICK_RES);
        brick_id %= (BRICK_RES * BRICK_RES);
        brick.y = brick_id / BRICK_RES;
        brick_id %= BRICK_RES;
        brick.x = brick_id;

        // printf("\nbrick %u: (%u,%u,%u)\n", brick_id, brick.x, brick.y, brick.z);

        for(unsigned int i = 0u; i < BRICK_VOXELS; i++)
        {
            unsigned int position_id = i;

            glm::uvec3 position = glm::uvec3(0u);
            position.z = position_id / (BRICK_VOXEL_DIM * BRICK_VOXEL_DIM);
            position_id %= (BRICK_VOXEL_DIM * BRICK_VOXEL_DIM);
            position.y = position_id / BRICK_VOXEL_DIM;
            position_id %= (BRICK_VOXEL_DIM);
            position.x = position_id;

            glm::uvec3 world = brick * BRICK_VOXEL_DIM + position;

            if(world.x >= VOLUME_VOXEL_DIM || world.y >= VOLUME_VOXEL_DIM || world.z >= VOLUME_VOXEL_DIM)
            {
                continue;
            }

            // printf("\nbrick %u, position %u: (%u,%u,%u)\n", occupied_bricks[idx], i, world.x, world.y, world.z);

            float2 data, ref;
            surf3Dread(&data, _volume_tsdf_data, world.x * sizeof(float2), world.y, world.z);
            surf3Dread(&ref, _volume_tsdf_ref, world.x * sizeof(float2), world.y, world.z);

            float2 fused;

            fused.y = ref.y + data.y;

            if(fused.y > 0.001f)
            {
                fused.x = data.x * data.y / fused.y + ref.x * ref.y / fused.y;
            }
            else
            {
                fused.x = data.y > ref.y ? data.x : ref.x;
            }

            surf3Dwrite(fused, _volume_tsdf_data, world.x * sizeof(float2), world.y, world.z);
        }
    }
}

__host__ void solve_for_h()
{
    int m = 0; // ED_GRAPH_NODE_COUNT * 15;
    int n = 0; // ED_GRAPH_NODE_COUNT * 15;

    hipblasHandle_t cublasHandle = nullptr;
    hipblasCreate(&cublasHandle);

    getLastCudaError("hipblasCreate failure");

    hipsparseHandle_t cusparseHandle = nullptr;
    hipsparseCreate(&cusparseHandle);

    getLastCudaError("hipsparseCreate failure");

    hipsparseMatDescr_t descr = nullptr;
    hipsparseCreateMatDescr(&descr);

    getLastCudaError("hipsparseCreateMatDescr failure");

    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    int *nnz_per_row_col = nullptr;
    int nnz_in_dev_memory;

    checkCudaErrors(hipMalloc((void **)&nnz_per_row_col, sizeof(int) * 2));

    hipsparseSnnz(cusparseHandle, HIPSPARSE_DIRECTION_ROW, m, n, descr, _jtj, m, nnz_per_row_col, &nnz_in_dev_memory);

    getLastCudaError("hipsparseSnnz failure");

    // printf ("\nNNZ: %u\n", nnz_in_dev_memory);

    hipDeviceSynchronize();

    float *csr_sorted_val_jtj = nullptr;
    int *csr_sorted_row_ptr_jtj = nullptr;
    int *csr_sorted_col_ind_jtj = nullptr;

    checkCudaErrors(hipMalloc((void **)&csr_sorted_val_jtj, sizeof(float) * nnz_in_dev_memory));
    checkCudaErrors(hipMalloc((void **)&csr_sorted_row_ptr_jtj, sizeof(int) * (m + 1)));
    checkCudaErrors(hipMalloc((void **)&csr_sorted_col_ind_jtj, sizeof(int) * nnz_in_dev_memory));

    hipsparseSdense2csr(cusparseHandle, m, n, descr, _jtj, m, nnz_per_row_col, csr_sorted_val_jtj, csr_sorted_row_ptr_jtj, csr_sorted_col_ind_jtj);

    getLastCudaError("hipsparseSdense2csr failure");

    hipDeviceSynchronize();

    float *h = (float *)malloc(sizeof(float) * n);
    float *rhs = (float *)malloc(sizeof(float) * n);

    for(int i = 0; i < n; i++)
    {
        rhs[i] = 1.0;
        h[i] = 0.0;
    }

    float *jtjh = nullptr;
    float *r = nullptr;
    float *p = nullptr;

    checkCudaErrors(hipMalloc((void **)&jtjh, n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&r, n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&p, n * sizeof(float)));

    checkCudaErrors(hipMemcpy(_h, h, n * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(r, rhs, n * sizeof(float), hipMemcpyHostToDevice));

    float alpha = 1.0f;
    float alpham1 = -1.0f;
    float beta = 0.0f;
    float r0 = 0.f;

    float a, b, na, r1, dot;

    const float tol = 1e-5f;

    hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz_in_dev_memory, &alpha, descr, csr_sorted_val_jtj, csr_sorted_row_ptr_jtj, csr_sorted_col_ind_jtj, _h, &beta, jtjh);

    hipblasSaxpy(cublasHandle, n, &alpham1, jtjh, 1, r, 1);
    hipblasSdot(cublasHandle, n, r, 1, r, 1, &r1);

    printf("initial residual = %e\n", sqrt(r1));

    int k = 1;

    while(r1 > tol * tol && k <= 10)
    {
        if(k > 1)
        {
            b = r1 / r0;
            hipblasSscal(cublasHandle, n, &b, p, 1);
            hipblasSaxpy(cublasHandle, n, &alpha, r, 1, p, 1);
        }
        else
        {
            hipblasScopy(cublasHandle, n, r, 1, p, 1);
        }

        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz_in_dev_memory, &alpha, descr, csr_sorted_val_jtj, csr_sorted_row_ptr_jtj, csr_sorted_col_ind_jtj, p, &beta, jtjh);
        hipblasSdot(cublasHandle, n, p, 1, jtjh, 1, &dot);
        a = r1 / dot;

        hipblasSaxpy(cublasHandle, n, &a, p, 1, _h, 1);
        na = -a;
        hipblasSaxpy(cublasHandle, n, &na, jtjh, 1, r, 1);

        r0 = r1;
        hipblasSdot(cublasHandle, n, r, 1, r, 1, &r1);
        hipDeviceSynchronize();
        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

    hipDeviceSynchronize();

    if(jtjh != nullptr)
    {
        checkCudaErrors(hipFree(jtjh));
    }

    if(r != nullptr)
    {
        checkCudaErrors(hipFree(r));
    }

    if(p != nullptr)
    {
        checkCudaErrors(hipFree(p));
    }

    if(csr_sorted_col_ind_jtj != nullptr)
    {
        checkCudaErrors(hipFree(csr_sorted_col_ind_jtj));
    }

    if(csr_sorted_row_ptr_jtj != nullptr)
    {
        checkCudaErrors(hipFree(csr_sorted_row_ptr_jtj));
    }

    if(csr_sorted_val_jtj != nullptr)
    {
        checkCudaErrors(hipFree(csr_sorted_val_jtj));
    }

    if(nnz_per_row_col != nullptr)
    {
        checkCudaErrors(hipFree(nnz_per_row_col));
    }

    if(rhs != nullptr)
    {
        free(rhs);
    }

    if(h != nullptr)
    {
        free(h);
    }

    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
}

extern "C" void init_cuda(glm::uvec3 &volume_res, struct_native_handles &native_handles)
{
    _volume_res = make_hipExtent(volume_res.x, volume_res.y, volume_res.z);

    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        if(deviceProperties.major >= 6 && deviceProperties.minor >= 1)
        {
            hipSetDevice(deviceIndex);
        }
    }

    size_t value;

    hipDeviceGetLimit(&value, hipLimitMallocHeapSize);
    printf("\n\nLimit Malloc Heap Size: %lu\n", value);

    hipDeviceGetLimit(&value, hipLimitStackSize);
    printf("\nLimit Stack Size: %lu\n", value);

    hipDeviceGetLimit(&value, cudaLimitDevRuntimeSyncDepth);
    printf("\nLimit Sync Depth: %lu\n", value);

    hipDeviceGetLimit(&value, cudaLimitDevRuntimePendingLaunchCount);
    printf("\nLimit Pending Launch: %lu\n\n", value);

    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cgr_buffer_vertex_counter, native_handles.buffer_vertex_counter, hipGraphicsRegisterFlagsReadOnly));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cgr_buffer_reference_mesh_vertices, native_handles.buffer_reference_vertices, hipGraphicsRegisterFlagsReadOnly));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cgr_buffer_bricks, native_handles.buffer_bricks, hipGraphicsRegisterFlagsReadOnly));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cgr_buffer_occupied, native_handles.buffer_occupied, hipGraphicsRegisterFlagsReadOnly));
    checkCudaErrors(hipGraphicsGLRegisterImage(&cgr_volume_tsdf_data, native_handles.volume_tsdf_data, GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    checkCudaErrors(hipGraphicsGLRegisterImage(&cgr_array2d_kinect_depths, native_handles.array2d_kinect_depths, GL_TEXTURE_2D_ARRAY, hipGraphicsRegisterFlagsReadOnly));

    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors(hipMalloc3DArray(&_volume_array_tsdf_ref, &channel_desc, _volume_res, hipArraySurfaceLoadStore));
}

extern "C" void deinit_cuda()
{
    checkCudaErrors(hipGraphicsUnregisterResource(cgr_buffer_vertex_counter));
    checkCudaErrors(hipGraphicsUnregisterResource(cgr_buffer_reference_mesh_vertices));
    checkCudaErrors(hipGraphicsUnregisterResource(cgr_buffer_bricks));
    checkCudaErrors(hipGraphicsUnregisterResource(cgr_buffer_occupied));
    checkCudaErrors(hipGraphicsUnregisterResource(cgr_volume_tsdf_data));
    checkCudaErrors(hipGraphicsUnregisterResource(cgr_array2d_kinect_depths));

    if(_ed_graph != nullptr)
    {
        checkCudaErrors(hipFree(_ed_graph));
    }

    if(_jtf != nullptr)
    {
        checkCudaErrors(hipFree(_jtf));
    }

    if(_jtj != nullptr)
    {
        checkCudaErrors(hipFree(_jtj));
    }

    if(_h != nullptr)
    {
        checkCudaErrors(hipFree(_h));
    }

    if(_volume_array_tsdf_ref != nullptr)
    {
        checkCudaErrors(hipFree(_volume_array_tsdf_ref));
    }
}

extern "C" void copy_reference_volume()
{
    checkCudaErrors(hipGraphicsMapResources(1, &cgr_volume_tsdf_data, 0));
    checkCudaErrors(hipGraphicsMapResources(1, &cgr_buffer_occupied, 0));

    hipArray *volume_array_tsdf_data = nullptr;
    checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&volume_array_tsdf_data, cgr_volume_tsdf_data, 0, 0));

    size_t occupied_brick_bytes;
    GLuint *brick_list;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&brick_list, &occupied_brick_bytes, cgr_buffer_occupied));

    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors(cudaBindSurfaceToArray(&_volume_tsdf_data, volume_array_tsdf_data, &channel_desc));
    checkCudaErrors(cudaBindSurfaceToArray(&_volume_tsdf_ref, _volume_array_tsdf_ref, &channel_desc));

    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel_copy_reference, 0, 0);

    unsigned max_bricks = ((unsigned)occupied_brick_bytes) / sizeof(unsigned);
    size_t gridSize = (max_bricks + blockSize - 1) / blockSize;

    kernel_copy_reference<<<gridSize, blockSize>>>(brick_list, max_bricks);

    getLastCudaError("render kernel failed");

    hipDeviceSynchronize();

    checkCudaErrors(hipGraphicsUnmapResources(1, &cgr_buffer_occupied, 0));
    checkCudaErrors(hipGraphicsUnmapResources(1, &cgr_volume_tsdf_data, 0));
}

extern "C" void sample_ed_nodes()
{
    if(_ed_graph != nullptr)
    {
        checkCudaErrors(hipFree(_ed_graph));
    }

    if(_jtf != nullptr)
    {
        checkCudaErrors(hipFree(_jtf));
    }

    if(_jtj != nullptr)
    {
        checkCudaErrors(hipFree(_jtj));
    }

    if(_h != nullptr)
    {
        checkCudaErrors(hipFree(_h));
    }

    //    checkCudaErrors(hipMalloc(&_jtj, ED_GRAPH_NODE_COUNT * 15 * ED_GRAPH_NODE_COUNT * 15 * sizeof(float)));
    //    checkCudaErrors(hipMalloc(&_jtf, ED_GRAPH_NODE_COUNT * 15 * sizeof(float)));
    //    checkCudaErrors(hipMalloc(&_h, ED_GRAPH_NODE_COUNT * 15 * sizeof(float)));
    //    checkCudaErrors(hipMalloc(&_ed_graph, ED_GRAPH_NODE_COUNT * sizeof(struct_ed_node)));

    size_t vx_bytes;
    GLuint *vx_counter;
    struct_vertex *vx_ptr;

    checkCudaErrors(hipGraphicsMapResources(1, &cgr_buffer_vertex_counter, 0));
    checkCudaErrors(hipGraphicsMapResources(1, &cgr_buffer_reference_mesh_vertices, 0));

    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&vx_counter, &vx_bytes, cgr_buffer_vertex_counter));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&vx_ptr, &vx_bytes, cgr_buffer_reference_mesh_vertices));

    // printf("\nvx_bytes: %zu\n", vx_bytes);

    kernel_sample_ed_nodes<<<dim3(4, 4, 4), dim3(2, 2, 2)>>>(vx_counter, vx_ptr, _ed_graph);

    getLastCudaError("render kernel failed");

    hipDeviceSynchronize();

    checkCudaErrors(hipGraphicsUnmapResources(1, &cgr_buffer_reference_mesh_vertices, 0));
    checkCudaErrors(hipGraphicsUnmapResources(1, &cgr_buffer_vertex_counter, 0));
}

extern "C" void align_non_rigid()
{
    hipArray *volume_array_tsdf_data = nullptr;

    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);

    size_t vx_bytes;
    GLuint *vx_counter;
    struct_vertex *vx_ptr;

    size_t brick_bytes;
    GLuint *brick_list;

    checkCudaErrors(hipGraphicsMapResources(1, &cgr_buffer_vertex_counter, 0));
    checkCudaErrors(hipGraphicsMapResources(1, &cgr_buffer_reference_mesh_vertices, 0));
    checkCudaErrors(hipGraphicsMapResources(1, &cgr_buffer_occupied, 0));
    checkCudaErrors(hipGraphicsMapResources(1, &cgr_volume_tsdf_data, 0));

    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&vx_counter, &vx_bytes, cgr_buffer_vertex_counter));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&vx_ptr, &vx_bytes, cgr_buffer_reference_mesh_vertices));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&brick_list, &brick_bytes, cgr_buffer_occupied));
    checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&volume_array_tsdf_data, cgr_volume_tsdf_data, 0, 0));

    checkCudaErrors(cudaBindSurfaceToArray(&_volume_tsdf_data, volume_array_tsdf_data, &channel_desc));
    checkCudaErrors(cudaBindSurfaceToArray(&_volume_tsdf_ref, _volume_array_tsdf_ref, &channel_desc));

    // kernel_jtj_jtf<<<dim3(8, 8, 8), dim3(4, 4, 4)>>>(_jtj, _jtf, vx_counter, vx_ptr, _ed_graph);

    getLastCudaError("render kernel failed");

    hipDeviceSynchronize();

    // solve_for_h();

    getLastCudaError("render kernel failed");

    hipDeviceSynchronize();

    unsigned max_bricks = ((unsigned)brick_bytes) / sizeof(unsigned);

    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel_copy_reference, 0, 0);

    size_t gridSize = (max_bricks + blockSize - 1) / blockSize;

    kernel_fuse_volume<<<gridSize, blockSize>>>(brick_list, max_bricks);

    getLastCudaError("render kernel failed");

    hipDeviceSynchronize();

    checkCudaErrors(hipGraphicsUnmapResources(1, &cgr_volume_tsdf_data, 0));
    checkCudaErrors(hipGraphicsUnmapResources(1, &cgr_buffer_occupied, 0));
    checkCudaErrors(hipGraphicsUnmapResources(1, &cgr_buffer_reference_mesh_vertices, 0));
    checkCudaErrors(hipGraphicsUnmapResources(1, &cgr_buffer_vertex_counter, 0));
}
